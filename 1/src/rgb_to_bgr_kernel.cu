#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "common.hpp"
#include "cuda_safe_call.hpp"
#include "rgb_to_bgr_cuda.hpp"


namespace {

__global__ void rgb_to_bgr_kernel(const std::uint8_t* const src, std::uint8_t* const dst,
                                  const std::size_t width, const std::size_t height) {
    const auto idx = blockDim.x * blockIdx.x + threadIdx.x;

    dst[idx * 3 + 0] = src[idx * 3 + 2];
    dst[idx * 3 + 1] = src[idx * 3 + 1];
    dst[idx * 3 + 2] = src[idx * 3 + 0];
}

}  // anonymous namespace


namespace cuda {

void rgb_to_bgr(const std::uint8_t* const src, std::uint8_t* const dst,
                const std::size_t width, const std::size_t height) {
    constexpr auto grid_dim = dim3{image_height};
    constexpr auto block_dim = dim3{image_width};
    const auto pixel_per_thread = static_cast<int>(
        ceilf(static_cast<float>(width * height) / (grid_dim.x * block_dim.x))
    );

    rgb_to_bgr_kernel<<<grid_dim, block_dim>>>(src, dst, width, height);
    CUDASafeCall();
}

}  // namespace cuda
