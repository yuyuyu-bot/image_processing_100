#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "common.hpp"
#include "cuda_safe_call.hpp"
#include "rgb_to_gray_cuda.hpp"


namespace {

__global__ void rgb_to_gray_kernel(const std::uint8_t* src, std::uint8_t* dst,
                                   const std::size_t width, const std::size_t height) {
    const auto idx = blockDim.x * blockIdx.x + threadIdx.x;

    constexpr auto r_coeff = 4899;
    constexpr auto g_coeff = 9617;
    constexpr auto b_coeff = 1868;
    constexpr auto normalize_shift_bits = 14;

    dst[idx] = (src[idx * 3 + 0] * r_coeff +
                src[idx * 3 + 1] * g_coeff +
                src[idx * 3 + 2] * b_coeff) >> normalize_shift_bits;
}

}  // anonymous namespace


namespace cuda {

void rgb_to_gray(const std::uint8_t* src, std::uint8_t* dst,
                 const std::size_t width, const std::size_t height) {
    constexpr auto grid_dim = dim3{image_height};
    constexpr auto block_dim = dim3{image_width};

    rgb_to_gray_kernel<<<grid_dim, block_dim>>>(src, dst, width, height);
    CUDASafeCall();
}

}  // namespace cuda
